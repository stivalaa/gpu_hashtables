#include "hip/hip_runtime.h"
/**********************************************************************************

 Test program for Lock-free hash table for CUDA; tested for CUDA 4.2 on 32-bit Ubuntu 10.10 and 64-bit Ubuntu 12.04.
 Developed at IIT Kanpur.

 This version changed by ADS to use the httslf hash table on GPU
 for comparison under identical conditions with the IIT Kanpur LockFreeHashTable

 Inputs: Percentage of add and delete operations (e.g., 30 50 for 30% add and 50% delete)
 Output: Prints the total time (in milliseconds) to execute the the sequence of operations

 Compilation flags: -O3 -arch sm_20 -I ~/NVIDIA_GPU_Computing_SDK/C/common/inc/ -DNUM_ITEMS=num_ops -DFACTOR=num_ops_per_thread 

 NUM_ITEMS is the total number of operations (mix of add, delete, search) to execute.

 FACTOR is the number of operations per thread.

 KEYS (cmomnad line parametr now)
 is the number of integer keys assumed in the range [10, 9+KEYS].
 The paper cited below states that the key range is [0, KEYS-1]. However, we have shifted the range by +10 so that
 the head sentinel key (the minimum key) can be chosen as zero. Any positive shift other than +10 would also work.

 The include path ~/NVIDIA_GPU_Computing_SDK/C/common/inc/ is needed for cutil.h.


 Related work:

 Prabhakar Misra and Mainak Chaudhuri. Performance Evaluation of Concurrent Lock-free Data Structures
 on GPUs. In Proceedings of the 18th IEEE International Conference on Parallel and Distributed Systems,
 December 2012.

 Stivala et al 2010 Lock-free parallel dynamic programming
 J Parallel Distrib Comput 70:389-848


 modified by Alex Stivala to include values so have key/value not just key
 $Id: iit_httslf_gpu_test.cu 4501 2013-01-01 09:18:30Z astivala $
***************************************************************************************/


#include <cassert> 
//#include"cutil.h"			// Comment this if cutil.h is not available
#include <cutil_inline.h>
#include"hip/hip_runtime.h"
#include"stdio.h"
#include <sys/time.h>

// Number of threads per block
#define NUM_THREADS 512

#include "httslf_gpu_kernel.cu"

#if __WORDSIZE == 64
typedef unsigned long long LL;
#else
typedef unsigned int LL;
#endif



// Supported operations
#define ADD (0)
#define DELETE (1)
#define SEARCH (2)



// The main kernel

__global__ void kernel(httslf_entry_t **hashtable, LL* items, LL* op, LL* result)
{
  // The array items holds the sequence of keys
  // The array op holds the sequence of operations
  // The array result, at the end, will hold the outcome of the operations
  // n points to an array of pre-allocated free linked list nodes

#ifdef USE_INSTRUMENT
  reset_counters();
#endif
  int tid,i;
  for(i=0;i<FACTOR;i++){		// FACTOR is the number of operations per thread
    tid=i*gridDim.x*blockDim.x+blockIdx.x*blockDim.x+threadIdx.x;
    if(tid>=NUM_ITEMS) return;

    // Grab the operation and the associated key and execute
    LL itm=items[tid];
    if(op[tid]==ADD){
      httslf_insert(hashtable, itm, itm+1);
      result[tid] = itm;
    }
    if(op[tid]==DELETE){
#ifdef ALLOW_DELETE
      httslf_delete(hashtable, itm);
#endif
      result[tid] = itm;
    }
    if(op[tid]==SEARCH){
      LL val;
      bool found=httslf_lookup(hashtable, itm, &val);
      if (found) {
        assert(val == itm+1); // compute capability 2.x has assert in device
      }
    }
  }
#ifdef USE_INSTRUMENT
  httslf_sumcounters();
#endif
}

/* Subtract the `struct timeval' values X and Y,
   storing the result in RESULT.
   Return 1 if the difference is negative, otherwise 0.  
(from GNU libc manual) */
     
static int
timeval_subtract (struct timeval *result, struct timeval *x, 
                  struct timeval *y)
{
  /* Perform the carry for the later subtraction by updating y. */
  if (x->tv_usec < y->tv_usec) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
    y->tv_usec -= 1000000 * nsec;
    y->tv_sec += nsec;
  }
  if (x->tv_usec - y->tv_usec > 1000000) {
    int nsec = (x->tv_usec - y->tv_usec) / 1000000;
    y->tv_usec += 1000000 * nsec;
    y->tv_sec -= nsec;
  }
     
  /* Compute the time remaining to wait.
     tv_usec is certainly positive. */
  result->tv_sec = x->tv_sec - y->tv_sec;
  result->tv_usec = x->tv_usec - y->tv_usec;
     
  /* Return 1 if result is negative. */
  return x->tv_sec < y->tv_sec;
}

int main(int argc, char** argv)
{
  unsigned int hTimer;
  double runtime;
  hipError_t rc;
  httslf_entry_t **devHashtable;
  httslf_entry_t **devCellpool;
  int i;
  struct timeval start_timeval,end_timeval,elapsed_timeval;  
  int etime;

  if (argc != 4) {
     printf("Need three arguments: keys, percent add ops and percent delete ops (e.g., 100000 30 50 for 100000 keys and 30%% add and 50%% delete).\nAborting...\n");
     exit(1);
  }

  int KEYS = atoi(argv[1]);
  int adds=atoi(argv[2]);
  int deletes=atoi(argv[3]);

  if (adds+deletes > 100) {
     printf("Sum of add and delete precentages exceeds 100.\nAborting...\n");
     exit(1);
  }

#ifndef ALLOW_DELETE
  if (deletes > 0) {
    fprintf(stderr, "compiled without ALLOW_DELETE: delete not supported\n");
    exit(1);
  }
#endif

  fprintf(stderr, "NUM_ITEMS = %d, KEYS = %d\n", NUM_ITEMS, KEYS);
  fprintf(stderr, "adds = %d, deletes = %d\n", adds, deletes);
  // Calculate the number of thread blocks
  // NUM_ITEMS = total number of operations to execute
  // NUM_THREADS = number of threads per block
  // FACTOR = number of operations per thread

  int blocks=(NUM_ITEMS%(NUM_THREADS*FACTOR)==0)?NUM_ITEMS/(NUM_THREADS*FACTOR):(NUM_ITEMS/(NUM_THREADS*FACTOR))+1;


  fprintf(stderr, "blocks = %d NUM_THREADS = %d\n", blocks, NUM_THREADS);
  fprintf(stderr, "NUM_ITEMS = %d FACTOR = %d\n", NUM_ITEMS, FACTOR);

   cutCreateTimer(&hTimer) ;
   cutResetTimer(hTimer) ;
   cutStartTimer(hTimer) ;


  gettimeofday(&start_timeval, NULL);
  /* allocate cell pool on device */
  size_t cell_size = sizeof(httslf_entry_t);
  int num_adds=(NUM_ITEMS*adds)/100;
  unsigned int devCellpool_num_items = num_adds+1;
  fprintf(stderr, "devCellpool_num_items = %u\n", devCellpool_num_items);
  size_t devCellpool_size = cell_size * devCellpool_num_items;


  // instead of cell pool, set heap size large enough to use device malloc
  // on compute cabability 2.x and higher
  // doulbe the size of the heap to allow plenty of space for malloc overhead
//  if ((rc = hipDeviceSetLimit(hipLimitMallocHeapSize, 2*devCellpool_size)) != hipSuccess)
//  {
//    fprintf(stderr, "hipDeviceSetLimit heap size failed %d (%s)\n",
//            rc, hipGetErrorString(rc));
//    exit(1);
//  }
//  size_t heapsize;
//  if ((rc = hipDeviceGetLimit(&heapsize, hipLimitMallocHeapSize))!=hipSuccess)
//  {
//    fprintf(stderr, "hipDeviceGetLimit heap size failed %d (%s)\n",
//            rc, hipGetErrorString(rc));
//  }
//  fprintf(stderr, "set cuda malloc heap size to %.1f MB\n", 
//          (double)heapsize/(1024*1024) );

  if ((rc = hipMalloc((void **)&devCellpool, devCellpool_size)) != hipSuccess)
  {
    fprintf(stderr, "hipMalloc devCellpool failed %d\n", rc);
    exit(1);
  }



  gettimeofday(&end_timeval, NULL);
  timeval_subtract(&elapsed_timeval, &end_timeval, &start_timeval);
  etime = 1000 * elapsed_timeval.tv_sec + elapsed_timeval.tv_usec/1000;
  fprintf(stderr, "hipMalloc %.1f MB cellpool elapsed time %d ms\n", 
          (double)devCellpool_size/(1024*1024), etime);

  /* set globals on device for clel pool alloc */
  if ((rc = hipMemcpyToSymbol(HIP_SYMBOL("cellpool"), &devCellpool, sizeof(httslf_entry_t *))) != hipSuccess)
  {
    fprintf(stderr, "hipMemcpyToSymbol cellpool failed %d\n", rc);
    exit(1);
  }

  /* set constanst on device for cell pool alloc */
  if ((rc = hipMemcpyToSymbol(HIP_SYMBOL("total_num_cells"), &devCellpool_num_items, sizeof(devCellpool_num_items))) != hipSuccess) 
  {
    fprintf(stderr, "hipMemcpyToSymbol poolsize failed%d\n",rc);
    exit(1);
  }
  

  gettimeofday(&start_timeval, NULL);
  /* allocate hashtable on device */
  if ((rc = hipMalloc((void **)&devHashtable, 
                       HTTSLF_SIZE*sizeof(httslf_entry_t *))) != hipSuccess)
  {
    fprintf(stderr, "hipMalloc devHashtable failed %d\n", rc);
    exit(1);
  }
  gettimeofday(&end_timeval, NULL);
  timeval_subtract(&elapsed_timeval, &end_timeval, &start_timeval);
  etime = 1000 * elapsed_timeval.tv_sec + elapsed_timeval.tv_usec/1000;
  fprintf(stderr, "hipMalloc %.1f MB hashtable elapsed time %d ms\n", 
          (double)HTTSLF_SIZE*sizeof(httslf_entry_t *)/(1024*1024), etime);

  gettimeofday(&start_timeval, NULL);
  /* set hashtable to all empty keys/values */
  httslf_reset<<<blocks, NUM_THREADS>>>(devHashtable);
  if ((rc = hipGetLastError()) != hipSuccess)
  {
    fprintf(stderr, "httslf_reset kernel error %d\n", rc);
  }
  cutilDeviceSynchronize();
  if ((rc = hipGetLastError()) != hipSuccess)
  {
    fprintf(stderr, "httslf_reset sync error %d\n", rc);
  }
  gettimeofday(&end_timeval, NULL);
  timeval_subtract(&elapsed_timeval, &end_timeval, &start_timeval);
  etime = 1000 * elapsed_timeval.tv_sec + elapsed_timeval.tv_usec/1000;
  fprintf(stderr, "httslf_reset elapsed time %d ms\n", etime);




  LL* op=(LL*)malloc(sizeof(LL)*NUM_ITEMS);		// Array of operations
  LL* items=(LL*)malloc(sizeof(LL)*NUM_ITEMS);		// Array of keys
  LL* result=(LL*)malloc(sizeof(LL)*NUM_ITEMS);		// Arrays of outcome
  srand(0);

  // NUM_ITEMS is the total number of operations to execute
  for(int i=0;i<NUM_ITEMS;i++){
    items[i]=10+rand()%KEYS;	// Keys
  }

  // Populate the op sequence
  for(i=0;i<(NUM_ITEMS*adds)/100;i++){
    op[i]=ADD;
  }
  for(;i<(NUM_ITEMS*(adds+deletes))/100;i++){
    op[i]=DELETE;
  }
  for(;i<NUM_ITEMS;i++){
    op[i]=SEARCH;
  }


   cutStopTimer(hTimer) ;
  fprintf(stderr, "host data generation time: %f ms\n", cutGetTimerValue(hTimer));

   cutResetTimer(hTimer) ;
   cutStartTimer(hTimer) ;

  // Allocate device memory

  LL* Citems;
  LL* Cop;
  LL* Cresult;
  
#ifdef _CUTIL_H_
  CUDA_SAFE_CALL(hipMalloc((void**)&Cresult, sizeof(LL)*NUM_ITEMS));
  CUDA_SAFE_CALL(hipMalloc((void**)&Citems, sizeof(LL)*NUM_ITEMS));
  CUDA_SAFE_CALL(hipMalloc((void**)&Cop, sizeof(LL)*NUM_ITEMS));
  CUDA_SAFE_CALL(hipMemcpy(Citems, items, sizeof(LL)*NUM_ITEMS, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(Cop, op, sizeof(LL)*NUM_ITEMS, hipMemcpyHostToDevice));
#else
  hipMalloc((void**)&Cresult, sizeof(LL)*NUM_ITEMS);
  hipMalloc((void**)&Citems, sizeof(LL)*NUM_ITEMS);
  hipMalloc((void**)&Cop, sizeof(LL)*NUM_ITEMS);
  hipMemcpy(Citems, items, sizeof(LL)*NUM_ITEMS, hipMemcpyHostToDevice);
  hipMemcpy(Cop, op, sizeof(LL)*NUM_ITEMS, hipMemcpyHostToDevice);
#endif

   cutStopTimer(hTimer) ;
  fprintf(stderr, "alloc and copy data (%f MB) to device time: %f ms\n", (double)sizeof(LL)*NUM_ITEMS*2/(1024*1024), cutGetTimerValue(hTimer));

   cutResetTimer(hTimer) ;
   cutStartTimer(hTimer) ;

  // Launch main kernel

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  
  kernel<<<blocks, NUM_THREADS>>>(devHashtable, Citems, Cop, Cresult);
  
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float time;
  hipEventElapsedTime(&time, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  // Print kernel execution time in milliseconds

  printf("elapsed time %lf\n",time);

  // Check for errors

  hipError_t error = hipGetLastError();
  if(hipSuccess!=error){
    printf("error:CUDA ERROR (%d) {%s}\n",error, hipGetErrorString(error));
    exit(-1);
  }

   cutStopTimer(hTimer) ;
  runtime =  cutGetTimerValue(hTimer);
  fprintf(stderr, "kernel exeuction time: %f ms\n", runtime);

   cutResetTimer(hTimer) ;
   cutStartTimer(hTimer) ;

  // Move results back to host memory

#ifdef _CUTIL_H_
  CUDA_SAFE_CALL(hipMemcpy(result, Cresult, sizeof(LL)*NUM_ITEMS, hipMemcpyDeviceToHost));
#else
  hipMemcpy(result, Cresult, sizeof(LL)*NUM_ITEMS, hipMemcpyDeviceToHost);
#endif

  cutStopTimer(hTimer) ;
  fprintf(stderr, "time to copy results back to host: %f ms\n", cutGetTimerValue(hTimer));


#ifdef USE_INSTRUMENT
  httslf_computestats<<<blocks, NUM_THREADS>>>(devHashtable);
  cutilDeviceSynchronize();
  if ((rc = hipGetLastError()) != hipSuccess)
  {
    fprintf(stderr, "httslf_computestats sync error %d (%s)\n", rc,hipGetErrorString(rc));
  }
  httslf_printstats<<<blocks, NUM_THREADS>>>();
  cutilDeviceSynchronize();
  if ((rc = hipGetLastError()) != hipSuccess)
  {
    fprintf(stderr, "httslf_printstats sync error %d (%s)\n", rc,hipGetErrorString(rc));
  }
#endif

  hipFree(devHashtable);
  cutilDeviceReset();

  return 0;
}
