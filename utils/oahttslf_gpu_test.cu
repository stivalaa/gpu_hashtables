#include "hip/hip_runtime.h"
/*****************************************************************************
 * 
 * File:    oahttslf_gpu_test.cu
 * Author:  Alex Stivala
 * Created: April 2009
 *
 * Test harness for open addressing thread-safe lock-free hash table.
 * 
 * Usage:
 *    oahttslf_gpu_test
 *
 * Preprocessor symbols:
 * DEBUG          - include extra assertion checks etc.
 *
 * $Id: oahttslf_gpu_test.cu 4556 2013-01-13 02:20:29Z astivala $
 *
 *****************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <pthread.h>
#include <sys/time.h>

#include "cutil_inline.h"
#include "hiprand/hiprand_kernel.h"

#define NUM_BLOCKS  128
#define NUM_THREADS 512

#include "oahttslf_gpu_kernel.cu"



#define NUM_INSERTIONS  100000


/*
 *TODO FIXME 
 *  this actually only uses the low 64 bits, doesn't even store high 
 */


typedef struct twothings {
  uint64_t high, low;
} SET;



/*****************************************************************************
 *
 * global functions (callable from host only)
 *
 *****************************************************************************/


/*
 * init_rng()
 *
 * Initialize CURAND pseudrandom number generator
 * See CUDA Toolkit 4.1 CURAND Guide (p.21)
 *
 * Parameters:
 *    state - CURAND state for random number generation
 *
 */
__global__ void init_rng(hiprandState *state)
{
  int tid=blockIdx.x*blockDim.x+threadIdx.x;

  /* give each therad same seed, different sequence number, no offset */
  hiprand_init(1234, tid, 0, &state[tid]);
}
 
 
__global__ void insert_random(oahttslf_entry_t *hashtable,
                               hiprandState *state)
{
  SET s,snew;
  int tid = blockIdx.x*blockDim.x+threadIdx.x;
  int num_threads = blockDim.x*gridDim.x;
  int my_num_insertions = NUM_INSERTIONS / num_threads;
  hiprandState localState = state[tid];/* cache state in fast local memory */
  uint64_t  value;
  uint64_t ivalue;
  int q;
#ifdef DEBUG
  int insertcount = 0, foundcount = 0;
#endif

#ifdef DEBUG
  printf("tid %d doing %d insertions\n", tid, my_num_insertions);
#endif

#ifdef USE_INSTRUMENT
  reset_counters();
#endif

  for (q = 0; q < my_num_insertions; q++)
  {
    s.low = ((uint32_t)hiprand(&localState) << 31 | (uint32_t)hiprand(&localState)) + 1;  
    if (s.low == 0)
        s.low = 1;
    s.high = 0;
    if (!oahttslf_lookup(hashtable, s.low, &value))
    {
      snew.low = ((uint32_t)hiprand(&localState) << 31 | (uint32_t)hiprand(&localState)) + 1;  
      if (snew.low == 0)
          snew.low = 1;
      snew.high = 0;
      if (oahttslf_lookup(hashtable, snew.low, &ivalue))
      {
#ifdef DEBUG
      foundcount++;
#endif
        if (ivalue != snew.low) {
          printf("ASSERTION FAILURE: thread %d: ivalue=%llX snew.low=%llX\n", tid , ivalue, snew.low);
          return;
        }
      }

      value = (uint64_t)s.low;
      oahttslf_insert(hashtable, s.low, value);
#ifdef DEBUG
      insertcount++;
#endif
    }
    else
    {
#ifdef DEBUG
      foundcount++;
#endif
    /*  assert(value == (uint64_t)s.low); */
      if (value != s.low) {
          printf("ASSERTION FAILURE 2: thread %d: value=%llX s.low=%llX\n",  tid, value, s.low);
          return;
      }
    }
  }
  state[tid] = localState; /* copy back new state from local cache */
#ifdef DEBUG
  printf("tid %d insertcount = %d foundcount = %d\n", tid,insertcount,foundcount);
#endif
#ifdef USE_INSTRUMENT
  oahttslf_sum_stats();
#endif
}




/*****************************************************************************
 *
 * static functions
 *
 *****************************************************************************/


/* Subtract the `struct timeval' values X and Y,
   storing the result in RESULT.
   Return 1 if the difference is negative, otherwise 0.  
(from GNU libc manual) */
     
int
timeval_subtract (struct timeval *result, struct timeval *x, 
                  struct timeval *y)
{
  /* Perform the carry for the later subtraction by updating y. */
  if (x->tv_usec < y->tv_usec) {
    int nsec = (y->tv_usec - x->tv_usec) / 1000000 + 1;
    y->tv_usec -= 1000000 * nsec;
    y->tv_sec += nsec;
  }
  if (x->tv_usec - y->tv_usec > 1000000) {
    int nsec = (x->tv_usec - y->tv_usec) / 1000000;
    y->tv_usec += 1000000 * nsec;
    y->tv_sec -= nsec;
  }
     
  /* Compute the time remaining to wait.
     tv_usec is certainly positive. */
  result->tv_sec = x->tv_sec - y->tv_sec;
  result->tv_usec = x->tv_usec - y->tv_usec;
     
  /* Return 1 if result is negative. */
  return x->tv_sec < y->tv_sec;
}


/***************************************************************************
 *
 * main
 *
 ***************************************************************************/

int main(int argc, char *argv[])
{
  int rc;
  struct timeval start_timeval,end_timeval,elapsed_timeval;  
  int etime;
  int blocks = NUM_BLOCKS;
  hiprandState *devStates;
  oahttslf_entry_t *devHashtable;


  gettimeofday(&start_timeval, NULL);
	// Pick the best GPU available, or if the developer selects one at the command line
	int devID = cutilChooseCudaDevice(argc, argv);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, devID);
	printf("> GPU Device has Compute Capabilities SM %d.%d\n\n", deviceProp.major, deviceProp.minor);
	int version = (deviceProp.major * 0x10 + deviceProp.minor);
	if (version < 0x20) {
    fprintf(stderr, "device with compute capability 2.0 or better is required\n");
    exit(1);
  }
  gettimeofday(&end_timeval, NULL);
  timeval_subtract(&elapsed_timeval, &end_timeval, &start_timeval);
  etime = 1000 * elapsed_timeval.tv_sec + elapsed_timeval.tv_usec/1000;
  fprintf(stderr, "device init (first CUDA call) elapsed time %d ms\n", etime);

  dim3 dimGrid(blocks)      ; // blocks
  dim3 dimBlock(NUM_THREADS); // threads

  fprintf(stderr, "Execution configuration: Grid = (%d,%d,%d) Block = (%d,%d,%d)\n", dimGrid.x,dimGrid.y,dimGrid.z, dimBlock.x,dimBlock.y,dimBlock.z);

  fprintf(stderr, "Doing %d insertions total with %d threads\n",
          NUM_INSERTIONS, dimGrid.x*dimBlock.x);

  gettimeofday(&start_timeval, NULL);

  /* allocate space on device for random number generator state */
  if ((rc = hipMalloc((void **)&devStates, 
                       blocks*NUM_THREADS*sizeof(hiprandState))) != hipSuccess)
  {
    fprintf(stderr, "hipMalloc devStates failed %d\n", rc);
    exit(1);
  }
  
  gettimeofday(&end_timeval, NULL);
  timeval_subtract(&elapsed_timeval, &end_timeval, &start_timeval);
  etime = 1000 * elapsed_timeval.tv_sec + elapsed_timeval.tv_usec/1000;
  fprintf(stderr, "alloc CURAND states (%.0f KB) elapsed time %d ms\n", 
          (float)blocks*NUM_THREADS*sizeof(hiprandState)/1024, etime);

  gettimeofday(&start_timeval, NULL);

  /* initialize device random number generator */
  init_rng<<<dimGrid, dimBlock>>>(devStates);
  if ((rc = hipGetLastError()) != hipSuccess)
  {
    fprintf(stderr, "init_rng kernel error %d\n", rc);
  }
  cutilDeviceSynchronize();
  if ((rc = hipGetLastError()) != hipSuccess)
  {
    fprintf(stderr, "init_rng sync error %d\n", rc);
  }
  gettimeofday(&end_timeval, NULL);
  timeval_subtract(&elapsed_timeval, &end_timeval, &start_timeval);
  etime = 1000 * elapsed_timeval.tv_sec + elapsed_timeval.tv_usec/1000;
  fprintf(stderr, "init CURAND kernel elapsed time %d ms\n", etime);

  gettimeofday(&start_timeval, NULL);
  /* allocate hashtable on device */
  if ((rc = hipMalloc((void **)&devHashtable, 
                       OAHTTSLF_SIZE*sizeof(oahttslf_entry_t))) != hipSuccess)
  {
    fprintf(stderr, "hipMalloc devHashtable failed %d\n", rc);
    exit(1);
  }
  gettimeofday(&end_timeval, NULL);
  timeval_subtract(&elapsed_timeval, &end_timeval, &start_timeval);
  etime = 1000 * elapsed_timeval.tv_sec + elapsed_timeval.tv_usec/1000;
  fprintf(stderr, "hipMalloc %.1f MB hashtable elapsed time %d ms\n", 
          (double)OAHTTSLF_SIZE*sizeof(oahttslf_entry_t)/(1024*1024), etime);

  gettimeofday(&start_timeval, NULL);
  /* set hashtable to all empty keys/values */
  oahttslf_reset<<<dimGrid, dimBlock>>>(devHashtable);
  if ((rc = hipGetLastError()) != hipSuccess)
  {
    fprintf(stderr, "oahttslf_reset kernel error %d\n", rc);
  }
  cutilDeviceSynchronize();
  if ((rc = hipGetLastError()) != hipSuccess)
  {
    fprintf(stderr, "oahttslf_reset sync error %d\n", rc);
  }
  gettimeofday(&end_timeval, NULL);
  timeval_subtract(&elapsed_timeval, &end_timeval, &start_timeval);
  etime = 1000 * elapsed_timeval.tv_sec + elapsed_timeval.tv_usec/1000;
  fprintf(stderr, "oahttslf_reset elapsed time %d ms\n", etime);


  gettimeofday(&start_timeval, NULL);

  /* Run the kernel */
  insert_random<<<dimGrid, dimBlock>>>(devHashtable, devStates);
  cutilDeviceSynchronize();
  if ((rc = hipGetLastError()) != hipSuccess)
  {
    fprintf(stderr, "insert_random sync error %d\n", rc);
  }

  gettimeofday(&end_timeval, NULL);
  timeval_subtract(&elapsed_timeval, &end_timeval, &start_timeval);
  etime = 1000 * elapsed_timeval.tv_sec + elapsed_timeval.tv_usec/1000;
  printf("elapsed time %d ms\n", etime);

#ifdef USE_INSTRUMENT
  oahttslf_print_stats<<<dimGrid, dimBlock>>>();
  cutilDeviceSynchronize();
  if ((rc = hipGetLastError()) != hipSuccess)
  {
    fprintf(stderr, "oahttslf_print_stats sync error %d\n", rc);
  }
#endif

  hipFree(devStates);
  hipFree(devHashtable);
  cutilDeviceReset();
  exit(EXIT_SUCCESS);
}

